#include "hip/hip_runtime.h"
#include <iostream>

#include "timer.hpp"

__global__ void saxpy(const float a, const float * x, float * y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    //int i = blockIdx.x + threadIdx.x * gridDim.x;
    y[i] = a * x[i] + y[i]; 
}

int main() {

for (int threads_per_block = 4; threads_per_block <= 1024; threads_per_block <<= 1) {

    int n = 1 << 25;
    //int threads_per_block = 256;
    int blocks_per_grid = n / threads_per_block;

    float a = 1.0;
    float * x;
    float * y;

    // allocate memory for the vectors
    hipMalloc(&x, sizeof(float) * n);
    hipMalloc(&y, sizeof(float) * n);

    timer stopwatch;

    stopwatch.start();
    saxpy<<< blocks_per_grid, threads_per_block >>>(a, x, y);
    hipDeviceSynchronize();
    stopwatch.stop();

    float time = stopwatch.elapsed();
    uint32_t num_bytes = n * sizeof(float) * 3; // 2 reads + 1 write

    std::cout << threads_per_block << " " <<  (num_bytes / time) * 1.0e-9f << std::endl;

    // deallocate memory for the vectors
    hipFree(x);
    hipFree(y);

}

}