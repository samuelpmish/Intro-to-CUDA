
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void long_kernel() {
    for (int i = 0; i < 1e2; ++i) {
        __nanosleep(1e6);
    }
}

float run_test(int num_blocks) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    long_kernel<<<num_blocks, 1>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);

    return duration;
}

int main(int argc, char* argv[]) {

    if (argc != 2) {
        printf("must specify number of blocks to run\n");
        exit(0);
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_SMs = prop.multiProcessorCount;
    int num_blocks = atoi(argv[1]);

    printf("number of blocks: %d, blocks / SM: %f, time = %f\n", 
        num_blocks, 
        float(num_blocks) / num_SMs,
        run_test(num_blocks)
    );

}