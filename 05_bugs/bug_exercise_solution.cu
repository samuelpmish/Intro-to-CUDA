#include "hip/hip_runtime.h"
#include <iostream>

#include "timer.hpp"

//  1
// ⌠   4        
// |  ---  dx = π
// ⌡  1+x²      
//  0 
__global__ void calculate_pi(int n, double * sum){

    // shared storage used to communicate between threads
    extern __shared__ double shmem[]; 

    auto f = [](double x) { return 4.0 / (1.0 + x * x); };

    double dx = 1.0 / n;
    int i = threadIdx.x + blockIdx.x * blockDim.x; 
    if (i < n) {
        double x = (i + 0.5) * dx;
        shmem[threadIdx.x] = f(x) * dx;
    } else {
        shmem[threadIdx.x] = 0.0;
    }

    __syncthreads();

    for(int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0){
        atomicAdd(sum, shmem[0]);
    }

}


int main() {

    static constexpr int n = 10'000'000;
    static constexpr double pi = 3.1415926535897932384626433832795028841971693993751058209749445923078164062862090;

    // allocate memory for the answer on the GPU, and initialize it to zero
    double * d_pi_approx;
    hipMalloc(&d_pi_approx, sizeof(double));
    hipMemset(d_pi_approx, 0, sizeof(double));

    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    int shmem = sizeof(double) * threads_per_block;
    float time_ms = 1000.0f * time([&](){
        calculate_pi<<< blocks_per_grid, threads_per_block, shmem >>>(n, d_pi_approx);
        hipDeviceSynchronize();
    });

    // copy the answer back from the GPU to compare
    double pi_approx;
    hipMemcpy(&pi_approx, d_pi_approx, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "computed pi ≈ " << pi_approx << " in " << time_ms << " ms, error " << fabs(pi - pi_approx) << std::endl;

    hipFree(d_pi_approx);

}