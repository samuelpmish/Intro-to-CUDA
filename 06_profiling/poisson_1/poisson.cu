#include "krylov.hpp"
#include "vector.hpp"
#include "timer.hpp"

#include <iostream>
#include <hipsparse.h>

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

hipsparseHandle_t handle = NULL;
hipsparseSpMatDescr_t matA;
hipsparseDnVecDescr_t vec_x, vec_y;
double alpha = 1.0;
double beta = 0.0;
void * d_buffer = nullptr;
size_t buffer_size = 0;

int * d_offsets;
int * d_columns;
double * d_values;
double * d_x;
double * d_y;

void initialize_CSR_matrix(int n) {

  std::vector< int > h_offsets(n * n * n + 1);

  std::vector< int > h_columns;
  h_columns.reserve(n * n * n * 7);

  std::vector< double > h_values;
  h_values.reserve(n * n * n * 7);

  auto id = [&](int x, int y, int z) {
    return z * n * n + y * n + x;
  };

  auto on_bdr = [&](int i) { 
    return (i == 0) || (i == (n-1)); 
  };

  auto push = [&](int col, double val) { 
    h_values.push_back(val);
    h_columns.push_back(col);
  };

  h_offsets[0] = 0;
  for (int k = 0; k < n; k++) {
    for (int j = 0; j < n; j++) {
      for (int i = 0; i < n; i++) {
        int cell_id = id(i, j, k);
        if (on_bdr(i) || on_bdr(j) || on_bdr(k)) {
          push(cell_id, 1.0);
        } else {
          push(cell_id, -6.0);
          push(id(i+1, j  , k  ), +1.0);
          push(id(i-1, j  , k  ), +1.0);
          push(id(i  , j+1, k  ), +1.0);
          push(id(i  , j-1, k  ), +1.0);
          push(id(i  , j  , k+1), +1.0);
          push(id(i  , j  , k-1), +1.0);
        }
        h_offsets[cell_id+1] = h_values.size();
      } 
    } 
  } 

  int ndof = n * n * n;
  int nnz = h_offsets.back();

  std::cout << "ndof: " << ndof << ", nnz: " << nnz << std::endl;

  hipMalloc(&d_offsets, (ndof + 1) * sizeof(int));
  hipMalloc(&d_columns, nnz * sizeof(int));
  hipMalloc(&d_values, nnz * sizeof(double));
  hipMalloc(&d_x, ndof * sizeof(double));
  hipMalloc(&d_y, ndof * sizeof(double));

  hipMemcpy(d_offsets, &h_offsets[0], (ndof + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_columns, &h_columns[0], nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values, &h_values[0], nnz * sizeof(double), hipMemcpyHostToDevice);

  CHECK_CUSPARSE(hipsparseCreate(&handle));
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA, ndof, ndof, nnz,
                                   d_offsets, d_columns, d_values,
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_x, ndof, d_x, HIP_R_64F));
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_y, ndof, d_y, HIP_R_64F));

  // allocate an external buffer if needed
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vec_x, &beta, vec_y, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &buffer_size))
  hipMalloc(&d_buffer, buffer_size);

}

void cleanup() {

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vec_x));
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vec_y));
  CHECK_CUSPARSE(hipsparseDestroy(handle));

  hipFree(d_offsets);
  hipFree(d_columns);
  hipFree(d_values);
  hipFree(d_x);
  hipFree(d_y);

}

int main() {

  int n = 256;
  int max_iterations = 100;
  double tolerance = 1.0e-2;

  initialize_CSR_matrix(n);

  auto A = [&](const gpu::vector & x){
    gpu::vector Ax(x.size()); 

    // update pointers and execute SpMV
    CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_x, x.ptr));
    CHECK_CUSPARSE(hipsparseDnVecSetValues(vec_y, Ax.ptr));
    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vec_x, &beta, vec_y, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, d_buffer)); 

    return Ax;
  };
  
  // solution = 0 on the boundary, and we 
  // have a Dirac delta source on the interior
  std::vector<double> rhs(n * n * n, 0.0);
  rhs[(n / 2) * n * n + (n / 2) * n + (n / 2)] = 1.0;
  gpu::vector b = rhs;

  timer stopwatch;

  stopwatch.start();
  gpu::vector x = cg(A, b, max_iterations, tolerance);
  hipDeviceSynchronize();
  stopwatch.stop();

  std::cout << "finished in " << stopwatch.elapsed() * 1000.0f << " ms" << std::endl;

  cleanup();

}